#include "hip/hip_runtime.h"
#include "ellipse.h"

__global__ void draw_ellipse_inefficient(u8* buffer, u32 size, int width, int height, u32 origin_x,
                                         u32 origin_y,int major, int minor, double angle, u8* image) {
    // prepare constants
    const double angle_cos = cos(angle);
    const double angle_sin = sin(angle);
    const double pow_major = pow(major, 2);
    const double pow_minor = pow(minor, 2);
    const u32 color_start = (origin_x + origin_y * width) * 3;

    // setup indexing
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // grid stride loop
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        // get 2d coordinates
        const int x = (i / 3) % width;
        const int y = (i / 3) / width;
        const int xc = x - int(origin_x);
        const int yc = y - int(origin_y);

        if ((pow((xc * angle_cos - yc * angle_sin), 2) / pow_major +
             pow((xc * angle_sin + yc * angle_cos), 2) / pow_minor) <= 1.0) {
            buffer[index] = image[color_start + (index % 3)];
        }
    }
}

__global__ void calc_fitness_inefficient(u8* buffer, u32 size, int width, int height, u32 origin_x,
                                         u32 origin_y,int major, int minor, double angle, u32 color,
                                         u8* image, u8* new_gen, FitCalcResult* result) {
    // prepare constants
    const double angle_cos = cos(angle);
    const double angle_sin = sin(angle);
    const double pow_major = pow(major, 2);
    const double pow_minor = pow(minor, 2);

    // setup indexing
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // grid stride loop
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < size; i += stride) {
        // get 2d coordinates
        const int x = (i / 3) % width;
        const int y = (i / 3) / width;
        const int xc = x - int(origin_x);
        const int yc = y - int(origin_y);

        u8 new_color;
        if ((pow((xc * angle_cos - yc * angle_sin), 2) / pow_major +
             pow((xc * angle_sin + yc * angle_cos), 2) / pow_minor) <= 1.0) {
            new_color = color >> ((index % 3) * 8);
        } else {
            new_color = new_gen[index];
        }

        result->new_score += abs(int(image[index]) - int(new_color));
        result->old_score += abs(int(image[index]) - int(new_gen[index]));
    }
}

void AllocBuffer(void** pointer, u32 size) {
    hipMallocManaged(pointer, size);
}

void FreeBuffer(void** pointer) {
    hipFree(pointer);
}

void DrawEllipseGPU(u8* buffer, u32 size, int width, int height, u32 origin_x, u32 origin_y,
                    int major, int minor, double angle, u8* image) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    draw_ellipse_inefficient<<<numBlocks, blockSize>>>(
        buffer, size, width, height, origin_x, origin_y, major, minor, angle, image);
    hipDeviceSynchronize();
}

void CalcFitnessGPU(u8* buffer, u32 size, int width, int height, u32 origin_x,
                              u32 origin_y,int major, int minor, double angle, u32 color,
                              u8* image, u8* new_gen, FitCalcResult* result) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    calc_fitness_inefficient<<<numBlocks, blockSize>>>(
        buffer, size, width, height, origin_x, origin_y, major, minor, angle, color, image,
        new_gen, result);
    hipDeviceSynchronize();
}